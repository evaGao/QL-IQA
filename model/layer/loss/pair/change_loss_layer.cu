#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
 
#include "caffe/blob.hpp"
#include "caffe/layer.hpp"
#include "caffe/proto/caffe.pb.h"

#include "caffe/layers/change_loss_layer.hpp"

#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void ChangeLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(1.0),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^1
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^1
      summer_vec_.gpu_data(),
      Dtype(0.0),
      dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin = this->layer_param_.change_loss_param().margin();
  Dtype add_front = this->layer_param_.change_loss_param().add_front(); 
  Dtype add_after = this->layer_param_.change_loss_param().add_after();
  Dtype range = this->layer_param_.change_loss_param().range();
  Dtype loss(0.0);
  Dtype dist(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int>(bottom[2]->cpu_data()[i])==0) {  // similar pairs
      dist=pow(std::max<Dtype>(-dist_sq_.cpu_data()[i],Dtype(0.0))-std::max<Dtype>(dist_sq_.cpu_data()[i]-margin,Dtype(0.0)),2)+pow(std::max<Dtype>(-bottom[0]->cpu_data()[i],Dtype(0.0))-std::max<Dtype>(bottom[0]->cpu_data()[i]-range,Dtype(0.0)),2);//损失为特征的距离,这里表示为dist
      loss=loss+dist;
    } else if(static_cast<int>(bottom[2]->cpu_data()[i])==1){  // sim标签为0
      dist=pow(std::max<Dtype>(-dist_sq_.cpu_data()[i],Dtype(0.0))-std::max<Dtype>(dist_sq_.cpu_data()[i]-2*margin,Dtype(0.0)),2)+pow(std::max<Dtype>(-bottom[0]->cpu_data()[i],Dtype(0.0))-std::max<Dtype>(bottom[0]->cpu_data()[i]-range,Dtype(0.0)),2);
      loss=loss+dist;
     } else if(static_cast<int>(bottom[2]->cpu_data()[i])==2){
        dist = pow(std::max<Dtype>(add_front-dist_sq_.cpu_data()[i],Dtype(0.0))-std::max<Dtype>(dist_sq_.cpu_data()[i]-add_after,Dtype(0.0)),2)+pow(std::max<Dtype>(-bottom[0]->cpu_data()[i],Dtype(0.0))-std::max<Dtype>(bottom[0]->cpu_data()[i]-range,Dtype(0.0)),2);
        loss =loss+dist;
      }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const Dtype alpha, const Dtype add_after, const Dtype add_front, const Dtype range,
    const Dtype* y, const Dtype* diff, const Dtype* dist_sq,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    Dtype dist=diff[i];
    Dtype mdist(0.0);
    Dtype mdistp(0.0);
    mdist=margin-dist;
    mdistp=2*margin-dist;
    if (static_cast<int>(y[n])==0) {  // similar pairs      		  
	if(dist<0.0){
		bottom_diff[i]=alpha*dist;
	}else if(mdist>=0.0){
		bottom_diff[i]=0;
	}else if(mdist<0.0){
			  bottom_diff[i]=-alpha*mdist;
	}
    } else if(static_cast<int>(y[n])==1) {  
		if(dist<0.0){
			  bottom_diff[i]=alpha*dist;
		  }else if(mdistp>=0){
			  bottom_diff[i]=0;
		  }else if(mdistp<0.0){
			  bottom_diff[i]=-alpha*mdistp;
		  }
        }else if(static_cast<int>(y[n])==2) {  
			if(dist<add_front){
			  bottom_diff[i]=-alpha*(add_front-dist);
		  }else if(dist<=add_after){
			  bottom_diff[i]=0;
		  }else if(dist>add_after){
			  bottom_diff[i]=-alpha*(add_after-dist);
		  }
        }
  }
}
template <typename Dtype>
__global__ void CLLBackward_one(const int count, const int channels,
    const Dtype margin, const Dtype alpha, const Dtype add_after, const Dtype add_front, const Dtype range,
    const Dtype* y, const Dtype* z, const Dtype* diff, const Dtype* dist_sq,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    Dtype dist=diff[i];
    Dtype mdist(0.0);
    Dtype mdistp(0.0);
    mdist=margin-dist;
    mdistp=2*margin-dist;
    if (static_cast<int>(y[n])==0) {  // similar pairs      		  
	if(dist<0.0){
		bottom_diff[i]=alpha*dist;
	}else if(mdist>=0.0){
		bottom_diff[i]=0;
	}else if(mdist<0.0){
			  bottom_diff[i]=-alpha*mdist;
	}
    } else if(static_cast<int>(y[n])==1) {  
		if(dist<0.0){
			  bottom_diff[i]=alpha*dist;
		  }else if(mdistp>=0){
			  bottom_diff[i]=0;
		  }else if(mdistp<0.0){
			  bottom_diff[i]=-alpha*mdistp;
		  }
        }else if(static_cast<int>(y[n])==2) {  
			if(dist<add_front){
			  bottom_diff[i]=-alpha*(add_front-dist);
		  }else if(dist<=add_after){
			  bottom_diff[i]=0;
		  }else if(dist>add_after){
			  bottom_diff[i]=-alpha*(add_after-dist);
		  }
        }
    if(z[n]<0.0){
		bottom_diff[i]=bottom_diff[i]+z[n];
	}else if(z[n]-range<=0.0){
		bottom_diff[i]=bottom_diff[i];
	}else if(z[n]-range>0.0){
		bottom_diff[i]=bottom_diff[i]+z[n]-range;
	}
		
  }
}

template <typename Dtype>
void ChangeLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
 // for (int i = 0; i < 2; ++i) {
    if (propagate_down[0]) {
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin = this->layer_param_.change_loss_param().margin();
      Dtype add_front = this->layer_param_.change_loss_param().add_front(); 
      Dtype add_after = this->layer_param_.change_loss_param().add_after();
      Dtype range = this->layer_param_.change_loss_param().range();
      const Dtype sign = (0 == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward_one<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, alpha,add_after,add_front,range,
          bottom[2]->gpu_data(),  // pair similarity 0 or 1
	  bottom[0]->gpu_data(),
          diff_.gpu_data(),  // the cached eltwise difference between a and b
          dist_sq_.gpu_data(),  // the cached square distance between a and b
          bottom[0]->mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
    }else if(propagate_down[1]){
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin = this->layer_param_.change_loss_param().margin();
      Dtype add_front = this->layer_param_.change_loss_param().add_front(); 
      Dtype add_after = this->layer_param_.change_loss_param().add_after();
      Dtype range = this->layer_param_.change_loss_param().range();
      const Dtype sign = (1 == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, alpha,add_after,add_front,range,
          bottom[2]->gpu_data(),  // pair similarity 0 or 1
          diff_.gpu_data(),  // the cached eltwise difference between a and b
          dist_sq_.gpu_data(),  // the cached square distance between a and b
          bottom[1]->mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
    }
 // }
  /*Dtype* bout1 = bottom[0]->mutable_gpu_diff();
  Dtype* bout2 = bottom[1]->mutable_gpu_diff();
  for(int i=0;i<bottom[0]->num();++i){
	bout1[i]=bout1[i]+bout2[i];
	bout2[i]=bout1[i];
  }*/
  //CUDA_POST_KERNEL_CHECK;
  
}
INSTANTIATE_LAYER_GPU_FUNCS(ChangeLossLayer);

}  // namespace caffe
